#include "hip/hip_runtime.h"
//#include "CudaKernels.h"

#include <iostream>
#include <hip/hip_vector_types.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#define N 10000

__global__ void add(int *a, int *b, int *c) {
	int tID = blockIdx.x;

	if (tID < N)
		c[tID] = a[tID] + b[tID];
}

void someOperation() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));

	// Fill Arrays
	for (int i = 0; i < N; i++) {
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
		std::cout<<a[i]<<"+"<< b[i]<<"="<<c[i]<<std::endl;

	// initialize all ten integers of a device_vector to 1
	thrust::device_vector<int> D(10, 1);
    // set the first seven elements of a vector to 9
   thrust::fill(D.begin(), D.begin() + 7, 9);
		    // initialize a host_vector with the first five elements of D
   thrust::host_vector<int> H(D.begin(), D.begin() + 5);
		      // set the elements of H to 0, 1, 2, 3, ...
	thrust::sequence(H.begin(), H.end());
					     // copy all of H back to the beginning of 
	thrust::copy(H.begin(), H.end(), D.begin());
								    // print D
	for(int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

/*
int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));

	// Fill Arrays
	for (int i = 0; i < N; i++) {
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	return 0;
}*/

