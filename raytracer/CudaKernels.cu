#include "hip/hip_runtime.h"
//#include "CudaKernels.h"

#include <iostream>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <math_functions.h>
//#include <cutil_math.h>


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#define N 10000

typedef struct {
    double3 p;
	 double r;
} SSphere;

typedef struct {
	double3 o;
	double3 d;
} SRay;

typedef struct {
	double tn;
	double tf;
	double l;
} SInInfo;

__device__ double3 minus(const double3& u, const double3& v) {
	return make_double3(u.x - v.x, u.y - v.y, u.z - v.z);
}

__device__ double dot(const double3& u, const double3& v) {
	return u.x*v.x + u.y*v.y + u.z*v.z;
}


__device__ int RaySphereIntersection(const SRay  &ray, const SSphere &sphere, SInInfo &t)
{
	double b, c, d;

	double3 sr = minus(ray.o, sphere.p);
	b =  dot(sr,ray.d);
	c = dot(sr,sr) - (sphere.r*sphere.r);
	d = b*b - c;
	if (d > 0) 
	{
		double e = sqrt(d);
		t.tn = -b-e;
		t.tf = -b+e;
		return 1;
	}
	return 0;
}


SSphere* dev_spheres;

__global__ void add(int *a, int *b, int *c) {
	int tID = blockIdx.x;

	if (tID < N)
		c[tID] = a[tID] + b[tID];
}

__global__ void hit(SSphere *s, int* r) {
	int tID = blockIdx.x;

	if(tID < 2) {
		if(s[tID].r == 2.0)
			r[tID] = 1;
		else
			r[tID] = 0;
	}
}

void loadUpScene() {
	SSphere spheres[2];
	spheres[0].p = make_double3(0.0, 0.0, 0.0);
	spheres[0].r = 2.0;
	spheres[1].p = make_double3(0.0, 0.0, 0.0);
	spheres[1].r = 2.0;


	int r[2];
	int* dev_r;

	int num = 2;

	hipMalloc((void **) &dev_spheres, num * sizeof(SSphere));
	hipMemcpy(dev_spheres, spheres, 2*sizeof(SSphere), hipMemcpyHostToDevice);

	hipMalloc((void **) &dev_r, 2 * sizeof(int));

	hit<<<N, 1>>>(dev_spheres, dev_r);

	hipMemcpy(r, dev_r, 2*sizeof(int), hipMemcpyDeviceToHost);

	std::cout<<r[0]<<" -- "<<r[1]<<std::endl;
}
void someOperation() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));

	// Fill Arrays
	for (int i = 0; i < N; i++) {
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
		std::cout<<a[i]<<"+"<< b[i]<<"="<<c[i]<<std::endl;

	// initialize all ten integers of a device_vector to 1
	thrust::device_vector<int> D(10, 1);
    // set the first seven elements of a vector to 9
   thrust::fill(D.begin(), D.begin() + 7, 9);
		    // initialize a host_vector with the first five elements of D
   thrust::host_vector<int> H(D.begin(), D.begin() + 5);
		      // set the elements of H to 0, 1, 2, 3, ...
	thrust::sequence(H.begin(), H.end());
					     // copy all of H back to the beginning of 
	thrust::copy(H.begin(), H.end(), D.begin());
								    // print D
	for(int i = 0; i < D.size(); i++)
		std::cout << "D[" << i << "] = " << D[i] << std::endl;
}

/*
int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));

	// Fill Arrays
	for (int i = 0; i < N; i++) {
		a[i] = i,
		b[i] = 1;
	}

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

	add<<<N,1>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

	return 0;
}*/

